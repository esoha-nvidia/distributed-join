#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include <vector>
#include <algorithm>
#include <memory>
#include <utility>
#include <tuple>
#include <string>
#include <stdexcept>
#include <cstdint>
#include <cstring>
#include <cstdlib>

#include <mpi.h>
#include <hip/hip_runtime_api.h>

#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "../src/topology.cuh"
#include "../src/communicator.h"
#include "../src/error.cuh"
#include "../src/generate_table.cuh"
#include "../src/distributed_join.cuh"

static std::string key_type = "int64_t";
static std::string payload_type = "int64_t";

static cudf::size_type BUILD_TABLE_NROWS_EACH_RANK = 100'000'000;
static cudf::size_type PROBE_TABLE_NROWS_EACH_RANK = 100'000'000;
static double SELECTIVITY = 0.3;
static bool IS_BUILD_TABLE_KEY_UNIQUE = true;
static int OVER_DECOMPOSITION_FACTOR = 1;
static bool USE_BUFFER_COMMUNICATOR = false;


void parse_command_line_arguments(int argc, char *argv[])
{
    for (int iarg = 0; iarg < argc; iarg++) {
        if (!strcmp(argv[iarg], "--key-type")) {
            key_type = argv[iarg + 1];
        }

        if (!strcmp(argv[iarg], "--payload-type")) {
            payload_type = argv[iarg + 1];
        }

        if (!strcmp(argv[iarg], "--build-table-nrows")) {
            BUILD_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
        }

        if (!strcmp(argv[iarg], "--probe-table-nrows")) {
            PROBE_TABLE_NROWS_EACH_RANK = atoi(argv[iarg + 1]);
        }

        if (!strcmp(argv[iarg], "--selectivity")) {
            SELECTIVITY = atof(argv[iarg + 1]);
        }

        if (!strcmp(argv[iarg], "--duplicate-build-keys")) {
            IS_BUILD_TABLE_KEY_UNIQUE = false;
        }

        if (!strcmp(argv[iarg], "--over-decomposition-factor")) {
            OVER_DECOMPOSITION_FACTOR = atoi(argv[iarg + 1]);
        }

        if (!strcmp(argv[iarg], "--use-buffer-communicator")) {
            USE_BUFFER_COMMUNICATOR = true;
        }
    }
}


void report_configuration()
{
    MPI_CALL( MPI_Barrier(MPI_COMM_WORLD) );

    int mpi_rank;
    int mpi_size;
    MPI_CALL( MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank) );
    MPI_CALL( MPI_Comm_size(MPI_COMM_WORLD, &mpi_size) );
    if (mpi_rank != 0)
        return;

    std::cout << "========== Parameters ==========" << std::endl;
    std::cout << std::boolalpha;
    std::cout << "Key type: " << key_type << std::endl;
    std::cout << "Payload type: " << payload_type << std::endl;
    std::cout << "Number of rows in the build table: "
              << static_cast<uint64_t>(BUILD_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6
              << " million" << std::endl;
    std::cout << "Number of rows in the probe table: "
              << static_cast<uint64_t>(PROBE_TABLE_NROWS_EACH_RANK) * mpi_size / 1e6
              << " million" << std::endl;
    std::cout << "Selectivity: " << SELECTIVITY << std::endl;
    std::cout << "Keys in build table are unique: " << IS_BUILD_TABLE_KEY_UNIQUE << std::endl;
    std::cout << "Over-decomposition factor: " << OVER_DECOMPOSITION_FACTOR << std::endl;
    std::cout << "Buffer communicator: " << USE_BUFFER_COMMUNICATOR << std::endl;
    std::cout << "================================" << std::endl;
}


int main(int argc, char *argv[])
{
    /* Initialize topology */

    setup_topology(argc, argv);

    /* Parse command line arguments */

    parse_command_line_arguments(argc, argv);
    report_configuration();

    cudf::size_type RAND_MAX_VAL = std::max(BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK) * 2;

    /* Initialize memory pool */

    size_t free_memory, total_memory;
    CUDA_RT_CALL(hipMemGetInfo(&free_memory, &total_memory));
    const size_t pool_size = free_memory - 5LL * (1LL << 29);  // free memory - 500MB

    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource();
    rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr {mr, pool_size, pool_size};
    rmm::mr::set_current_device_resource(&pool_mr);

    /* Initialize communicator */

    int mpi_rank;
    int mpi_size;
    MPI_CALL( MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank) );
    MPI_CALL( MPI_Comm_size(MPI_COMM_WORLD, &mpi_size) );

    UCXCommunicator* communicator = initialize_ucx_communicator(
        USE_BUFFER_COMMUNICATOR, 2 * mpi_size, 800'000'000LL / mpi_size - 100'000LL
    );

    /* Generate build table and probe table on each node */

    std::unique_ptr<cudf::table> left;
    std::unique_ptr<cudf::table> right;

    #define generate_tables(KEY_T, PAYLOAD_T)                                  \
    {                                                                          \
        std::tie(left, right) = generate_tables_distributed<KEY_T, PAYLOAD_T>( \
            BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK,          \
            SELECTIVITY, RAND_MAX_VAL, IS_BUILD_TABLE_KEY_UNIQUE,              \
            communicator                                                       \
        );                                                                     \
    }

    #define generate_tables_key_type(KEY_T)                                    \
    {                                                                          \
        if (payload_type == "int64_t") {                                       \
            generate_tables(KEY_T, int64_t)                                    \
        } else if (payload_type == "int32_t") {                                \
            generate_tables(KEY_T, int32_t)                                    \
        } else {                                                               \
            throw std::runtime_error("Unknown payload type");                  \
        }                                                                      \
    }

    if (key_type == "int64_t") {
        generate_tables_key_type(int64_t)
    } else if (key_type == "int32_t") {
        generate_tables_key_type(int32_t)
    } else {
        throw std::runtime_error("Unknown key type");
    }

    /* Distributed join */

    CUDA_RT_CALL(hipDeviceSynchronize());

    MPI_Barrier(MPI_COMM_WORLD);
    hipProfilerStart();
    double start = MPI_Wtime();

    std::unique_ptr<cudf::table> join_result = distributed_inner_join(
        left->view(), right->view(),
        {0}, {0}, {std::pair<cudf::size_type, cudf::size_type>(0, 0)},
        communicator, OVER_DECOMPOSITION_FACTOR
    );

    MPI_Barrier(MPI_COMM_WORLD);
    double stop = MPI_Wtime();
    hipProfilerStop();

    if (mpi_rank == 0) {
        std::cout << "Elasped time (s) " << stop - start << std::endl;
    }

    /* Cleanup */

    communicator->finalize();
    delete communicator;

    return 0;
}
