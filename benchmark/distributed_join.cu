/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include <vector>
#include <algorithm>
#include <memory>
#include <utility>
#include <tuple>
#include <cstdint>
#include <cstring>
#include <cstdlib>

#include <mpi.h>
#include <hip/hip_runtime_api.h>

#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <rmm/mr/device/default_memory_resource.hpp>
#include <rmm/mr/device/cnmem_memory_resource.hpp>

#include "../src/topology.cuh"
#include "../src/communicator.h"
#include "../src/error.cuh"
#include "../src/generate_table.cuh"
#include "../src/distributed_join.cuh"

#define KEY_T int64_t
#define PAYLOAD_T int64_t

static cudf::size_type BUILD_TABLE_NROWS_EACH_RANK = 100'000'000;
static cudf::size_type PROBE_TABLE_NROWS_EACH_RANK = 100'000'000;
static double SELECTIVITY = 0.3;
static KEY_T RAND_MAX_VAL = 200'000'000;
static bool IS_BUILD_TABLE_KEY_UNIQUE = true;
static int OVER_DECOMPOSITION_FACTOR = 1;
static bool USE_BUFFER_COMMUNICATOR = false;


int main(int argc, char *argv[])
{
    /* Initialize topology */

    setup_topology(argc, argv);

    /* Initialize memory pool */

    size_t free_memory, total_memory;
    CUDA_RT_CALL(hipMemGetInfo(&free_memory, &total_memory));
    const size_t pool_size = free_memory - 5LL * (1LL << 29);  // free memory - 500MB

    rmm::mr::cnmem_memory_resource cnmem_mr {pool_size};
    rmm::mr::set_default_resource(&cnmem_mr);

    /* Initialize communicator */

    int mpi_rank;
    int mpi_size;
    MPI_CALL( MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank) );
    MPI_CALL( MPI_Comm_size(MPI_COMM_WORLD, &mpi_size) );

    UCXCommunicator* communicator = initialize_ucx_communicator(
        USE_BUFFER_COMMUNICATOR, 2 * mpi_size, 800'000'000LL / mpi_size - 100'000LL
    );

    /* Generate build table and probe table on each node */

    std::unique_ptr<cudf::table> left;
    std::unique_ptr<cudf::table> right;

    std::tie(left, right) = generate_tables_distributed<KEY_T, PAYLOAD_T>(
        BUILD_TABLE_NROWS_EACH_RANK, PROBE_TABLE_NROWS_EACH_RANK,
        SELECTIVITY, RAND_MAX_VAL, IS_BUILD_TABLE_KEY_UNIQUE,
        communicator
    );

    /* Distributed join */

    CUDA_RT_CALL(hipDeviceSynchronize());

    MPI_Barrier(MPI_COMM_WORLD);
    hipProfilerStart();
    double start = MPI_Wtime();

    std::unique_ptr<cudf::table> join_result = distributed_inner_join(
        left->view(), right->view(),
        {0}, {0}, {std::pair<cudf::size_type, cudf::size_type>(0, 0)},
        communicator, OVER_DECOMPOSITION_FACTOR
    );

    MPI_Barrier(MPI_COMM_WORLD);
    double stop = MPI_Wtime();
    hipProfilerStop();

    if (mpi_rank == 0) {
        std::cout << "Elasped time (s) " << stop - start << std::endl;
    }

    /* Cleanup */

    communicator->finalize();
    delete communicator;

    return 0;
}
